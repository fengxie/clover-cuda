#include "hip/hip_runtime.h"

#include <cinttypes>

#include <hip/hip_runtime.h>

#include <png.h>

__global__ void clover_kernel(uchar4 *image, size_t stride, const dim2 center);
extern int write_png(char* filename, int width, int height, unsigned char *buffer, char* title);

int main(const int argc, const char *argv[])
{
    unsigned char *output;
    unsigned char *device_output;

    const size_t height = 1024;
    const size_t width  = 1024;
    const size_t size   = 4 * height * width;
    output = new uchar[size];

    CUDA_ASSERT(hipMalloc((void **)&device_output, size * sizeof(uchar)), "");

    CUDA_ASSERT(hipMemcpy(device_output, output, size * sizeof(uchar), hipMemcpyHostToDevice), "");

    dim3 blocks(1, 1, 1);

    CUDA_ASSERT(hipMemcpy(output, device_output, size * sizeof(uchar), hipMemcpyDeviceToHost), "");
    write_png("clover.png", width, height, output, NULL);

    CUDA_ASSERT(hipFree(device_output), "");

    delete[] output;

    return 0;
}

int write_png(char* filename, int width, int height, unsigned char *buffer, char* title)
{
   int code = 0;
   FILE *fp = NULL;
   png_structp png_ptr = NULL;
   png_infop info_ptr = NULL;
   png_bytep row = NULL;

   // Open file for writing (binary mode)
   fp = fopen(filename, "wb");
   if (fp == NULL) {
      fprintf(stderr, "Could not open file %s for writing\n", filename);
      code = 1;
      goto finalise;
   }

   // Initialize write structure
   png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
   if (png_ptr == NULL) {
      fprintf(stderr, "Could not allocate write struct\n");
      code = 1;
      goto finalise;
   }

   // Initialize info structure
   info_ptr = png_create_info_struct(png_ptr);
   if (info_ptr == NULL) {
      fprintf(stderr, "Could not allocate info struct\n");
      code = 1;
      goto finalise;
   }

   // Setup Exception handling
   if (setjmp(png_jmpbuf(png_ptr))) {
      fprintf(stderr, "Error during png creation\n");
      code = 1;
      goto finalise;
   }

   png_init_io(png_ptr, fp);

   // Write header (8 bit colour depth)
   png_set_IHDR(png_ptr, info_ptr, width, height,
         8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
         PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

   // Set title
   if (title != NULL) {
      png_text title_text;
      title_text.compression = PNG_TEXT_COMPRESSION_NONE;
      title_text.key = "Title";
      title_text.text = title;
      png_set_text(png_ptr, info_ptr, &title_text, 1);
   }

   png_write_info(png_ptr, info_ptr);

   // Write image data
   int x, y;
   for (y = 0 ; y < height ; y++) {
      png_write_row(png_ptr, buffer + y * width * 4);
   }

   // End write
   png_write_end(png_ptr, NULL);

finalise:
   if (fp != NULL) fclose(fp);
   if (info_ptr != NULL) png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
   if (png_ptr != NULL) png_destroy_write_struct(&png_ptr, (png_infopp)NULL);

   return code;
}